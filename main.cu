#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// this is the deleayed learning, which gives an effect similar to batching
#define BATCHING 512

// this pushes a prepared sense data into a subspace on device
// often this will be prepared on the host
__global__ void pushSenseData() {
}

// this pulls a calculated sense data from a subspace on device
// often this will be displayed on the host
__global__ void pullSenseData() {
}

// this create blank sense data for a subspace on device
// not sure if this makes sense
__global__ void blankSenseData() {
}

// this pushes prepared states into short time (state) memory on device
__global__ void pushStates() {
}

// this pulls calculated states from short time (state) memory on device
__global__ void pullStates() {
}

// this create blank states for short time (state) memory on device
__global__ void blankStates() {
}

// this pushes a weight tensor for the complete manifold on device
// should perhaps be segmented
__global__ void pushWeights() {
}

// this pulls a weight tensor for the complete manifold on device
// should perhaps be segmented
__global__ void pullWeights() {
}

// this create a blank weight tensor for the complete manifold on device
__global__ void blankWeights() {
}

// Evaluate apical dendrites and set firing of synapses
// current activation of synapses are kept for later
__global__ void activateApicalDendrite() {
    // activateSynapses();
    // tentativeLearnSynapses();
}

// Evaluate basal dendrites and set firing of synapses
// current activation of synapses are kept for later
__global__ void activateBasalDendrite(int layer) {
    // activateSynapses();
    // tentativeLearnSynapses();
}

// Evaluate dendrites and set tentative learning states of synapses
__global__ void spikeNode(int layer) {
    // previousActivation = currentActivation;
    // activateAxon();
}

int main() {
    // declare all variables
    unsigned int = 0;
    bool die = false;
    
    // allocate host memory
    // malloc();
    
    // allocate device memory
    // hipMalloc();
    
    // load weights from store on host
    //

    // load state from store on host
    
    while (!die) {
        tick++;
        
        // push sense data from host to device
        // pushSenseData();
        
        //-- neo --
        // updates after sense data
        
        // evaluate apical state on device
        // activateApicalDendrite<<<numblocks, numthreads>>>();
        
        // evaluate each of the layers
        // activateBasalDendrite<<<numblocks, numthreads>>>(1);
        // spikeNode<<<numblocks, numthreads>>>(1);
        // activateBasalDendrite<<<numblocks, numthreads>>>(2);
        // spikeNode<<<numblocks, numthreads>>>(2);
        // activateBasalDendrite<<<numblocks, numthreads>>>(3);
        // spikeNode<<<numblocks, numthreads>>>(3);
        // activateBasalDendrite<<<numblocks, numthreads>>>(4);
        // spikeNode<<<numblocks, numthreads>>>(4);
        // activateBasalDendrite<<<numblocks, numthreads>>>(5);
        // spikeNode<<<numblocks, numthreads>>>(5);
        
        //-- basket --
        // should this go first?
        // activateBasket<<<numblocks, numthreads>>>();
        
        //-- candle --
        // updates after neo
        // activateCandle<<<numblocks, numthreads>>>();
        
        //-- learn --
        // updates after candle
        if (tick % BATCHING) {
            tick = 0;
            //learnSynapses<<numblocks, numthreads>>();
        }
        
        //-- state --
        // updates after candle
        
        //-- assoc --
        // updates after candle
        
        // pull sense data from device to host
        // pullSenseData();
    }
    
    // save state to store on host
    // pullState();
        
    // free host memory
    // hipFree();
        
    // free device memory
    // free();
    
}